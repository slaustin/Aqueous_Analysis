#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define CHECK(call) {   const hipError_t error = call; if (error != hipSuccess) { printf("Error: %s:%d, ", __FILE__, __LINE__); printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); exit(1); }}

__global__ void countagua (int *a,int *b,int *c,int na,int nb,float *xx,float *yy,float *zz,int *convers,int natim,int x_min,int y_min,int z_min,int x_max,int y_max,int z_max,int x_side,int y_side,int z_side,int *space,int num_points,int num_protons) {

        int k=threadIdx.x + blockDim.x * blockIdx.x;
        int r=blockIdx.x;
        int kk,j,jj,q,xxx,yyy,zzz,rx,ry,rz;
        long int n_of_grid;
        float curr_dist,dist_a;

	if (k<na){
		kk=a[k]-1;
		for(j=0;j<nb;j++){
			jj=b[j]-1;
			if(kk != jj){
				if(convers[jj] == 1){
					curr_dist=0.0;
					curr_dist=sqrtf(((xx[kk]-xx[jj])*(xx[kk]-xx[jj]))+((yy[kk]-yy[jj])*(yy[kk]-yy[jj]))+((zz[kk]-zz[jj])*(zz[kk]-zz[jj])));
					if(curr_dist < 3.5){
						//Proton Loop
						for(q=1;q<=num_protons;q++){
							jj=b[j+q]-1;
							dist_a=0.0;
							dist_a=sqrtf(((xx[kk]-xx[jj])*(xx[kk]-xx[jj]))+((yy[kk]-yy[jj])*(yy[kk]-yy[jj]))+((zz[kk]-zz[jj])*(zz[kk]-zz[jj])));
							if(dist_a < 2.45){
								//Acceptor
								if ((xx[kk] >= x_min)&&(yy[kk] >= y_min)&&(zz[kk] >= z_min)&&(xx[kk] <= x_max)&&(yy[kk] <= y_max)&&(zz[kk] <= z_max)){
									c[k]+=1;}
								xxx=floor(xx[kk]);
								yyy=floor(yy[kk]);
								zzz=floor(zz[kk]);
								if ((xxx >= x_min)&&(yyy >= y_min)&&(zzz >= z_min)&&(xxx <= x_max)&&(yyy <= y_max)&&(zzz <= z_max)){
									for(rx=xxx-1;rx<xxx+2;rx++){
									for(ry=yyy-1;ry<yyy+2;ry++){
									for(rz=zzz-1;rz<zzz+2;rz++){
									if ((rx >= x_min)&&(ry >= y_min)&&(rz >= z_min)&&(rx <= x_max)&&(ry <= y_max)&&(rz <= z_max)){
										n_of_grid=(rx-x_min)+((ry-y_min)*x_side)+((rz-z_min)*x_side*y_side)+(r*x_side*y_side*z_side);
										if((n_of_grid >= 0)&&(n_of_grid < num_points)){space[n_of_grid]+=1;}
									}
									}
									}
									}
								}
								//Donor
								if ((xx[jj] >= x_min)&&(yy[jj] >= y_min)&&(zz[jj] >= z_min)&&(xx[jj] <= x_max)&&(yy[jj] <= y_max)&&(zz[jj] <= z_max)){
									c[k]+=1;}
								xxx=floor(xx[jj]);
								yyy=floor(yy[jj]);
								zzz=floor(zz[jj]);
								if ((xxx >= x_min)&&(yyy >= y_min)&&(zzz >= z_min)&&(xxx <= x_max)&&(yyy <= y_max)&&(zzz <= z_max)){
									for(rx=xxx-1;rx<xxx+2;rx++){
									for(ry=yyy-1;ry<yyy+2;ry++){
									for(rz=zzz-1;rz<zzz+2;rz++){
									if ((rx >= x_min)&&(ry >= y_min)&&(rz >= z_min)&&(rx <= x_max)&&(ry <= y_max)&&(rz <= z_max)){
										n_of_grid=(rx-x_min)+((ry-y_min)*x_side)+((rz-z_min)*x_side*y_side)+(r*x_side*y_side*z_side);
										if((n_of_grid >= 0)&&(n_of_grid < num_points)){space[n_of_grid]+=1;}
									}
									}
									}
									}
								}
							}
						}
					}
				}
			}
		}
	}

} // End of Global

extern "C" void water_wrapper_(int *frame_count, double *x,double *y,double *z,int *min_x,int *min_y,int *min_z,int *max_x, int *max_y, int *max_z,int *sidex,int *sidey,int *sidez, int *grid_array,int *natoma,int *lista,int *natomb,int *listb,int *conversion,int *natim, int *prot_num, int *curr_count)
{//main

int k,r,rr,rrr,t,blocks,threads,sum;
long int num_points,num_points2,n_of_grid,n_of_grid2;
int devCount;
int num_atom_a=*natoma;
int num_atom_b=*natomb;
int n_atim=*natim;
int minn_x=*min_x;
int minn_y=*min_y;
int minn_z=*min_z;
int maxx_x=*max_x;
int maxx_y=*max_y;
int maxx_z=*max_z;
int x_side=*sidex;
int y_side=*sidey;
int z_side=*sidez;
int num_protons=*prot_num;
int *local_lista, *local_listb, *local_listc, *local_conversion, *local_space, *local_space2;
float *local_x, *local_y, *local_z;
int *dev_lista, *dev_listb, *dev_listc, *dev_conversion, *dev_space;
float *dev_x,*dev_y,*dev_z;

/////////////////////////////////////////////////////////////////

hipGetDeviceCount(&devCount);

for (int i = 0; i < devCount; ++i){
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        threads=devProp.maxThreadsPerBlock;
}

blocks=0;
num_points=0;
num_points2=0;
blocks=ceil(float(num_atom_a)/float(threads))+1;
num_points=x_side*y_side*z_side*blocks;
num_points2=x_side*y_side*z_side;

local_lista=(int *)malloc(num_atom_a*sizeof(int));
if(local_lista == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_listb=(int *)malloc(num_atom_b*sizeof(int));
if(local_listb == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_listc=(int *)malloc(num_atom_a*sizeof(int));
if(local_listc == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_x=(float *)malloc(n_atim*sizeof(float));
if(local_x == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_y=(float *)malloc(n_atim*sizeof(float));
if(local_y == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_z=(float *)malloc(n_atim*sizeof(float));
if(local_z == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_conversion=(int *)malloc(n_atim*sizeof(int));
if(local_conversion == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_space=(int *)malloc(num_points*sizeof(int));
if(local_space == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_space2=(int *)malloc(num_points2*sizeof(int));
if(local_space2 == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

memset(local_space,0,num_points*sizeof(int));
memset(local_space2,0,num_points2*sizeof(int));
memset(local_listc,0,num_atom_a*sizeof(int));
///////////////////////////////////////////////////////


// Transfer Pointers from Fortran to Local C Arrays
for (k=0;k<num_atom_a;k++){
     local_lista[k]=lista[k];
}

for (k=0;k<num_atom_b;k++){
     local_listb[k]=listb[k];
}

for (k=0;k<n_atim;k++){
     local_x[k]=x[k];
     local_y[k]=y[k];
     local_z[k]=z[k];
     local_conversion[k]=conversion[k];
}

///////////////////////////////////////////////////////

CHECK (hipMalloc((void **) &dev_lista, num_atom_a*sizeof(int)) );
CHECK (hipMalloc((void **) &dev_listb, num_atom_b*sizeof(int)) );
CHECK (hipMalloc((void **) &dev_listc, num_atom_a*sizeof(int)) );
CHECK (hipMalloc((void **) &dev_space, num_points*sizeof(int)) );
CHECK (hipMalloc((void **) &dev_x, n_atim*sizeof(float)) );
CHECK (hipMalloc((void **) &dev_y, n_atim*sizeof(float)) );
CHECK (hipMalloc((void **) &dev_z, n_atim*sizeof(float)) );
CHECK (hipMalloc((void **) &dev_conversion, n_atim*sizeof(int)) );


CHECK (hipMemcpy(dev_x, local_x, n_atim*sizeof(float), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_y, local_y, n_atim*sizeof(float), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_z, local_z, n_atim*sizeof(float), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_lista, local_lista, num_atom_a*sizeof(int), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_listb, local_listb, num_atom_b*sizeof(int), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_listc, local_listc, num_atom_a*sizeof(int), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_conversion, local_conversion, n_atim*sizeof(int), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_space, local_space, num_points*sizeof(int), hipMemcpyHostToDevice) );

///////////////////////////////////////////////////////

//printf("Launch!\n");
//printf("Blocks=%i\n",blocks);
//printf("Threads=%i\n",threads);

countagua<<<blocks,threads>>>(dev_lista,dev_listb,dev_listc,num_atom_a,num_atom_b,dev_x,dev_y,dev_z,dev_conversion,n_atim,minn_x,minn_y,minn_z,maxx_x,maxx_y,maxx_z,x_side,y_side,z_side,dev_space,num_points,num_protons);

CHECK (hipMemcpy(local_space, dev_space, num_points*sizeof(int), hipMemcpyDeviceToHost) );
CHECK (hipMemcpy(local_listc, dev_listc, num_atom_a*sizeof(int), hipMemcpyDeviceToHost) );

CHECK (hipFree(dev_lista) ); 
CHECK (hipFree(dev_listb) ); 
CHECK (hipFree(dev_listc) ); 
CHECK (hipFree(dev_x) ); 
CHECK (hipFree(dev_y) ); 
CHECK (hipFree(dev_z) ); 
CHECK (hipFree(dev_conversion) ); 
CHECK (hipFree(dev_space) );
CHECK (hipDeviceReset());

///////////////////////////////////////////////////////

n_of_grid=0;
n_of_grid2=0;
sum=0;

for (k=0;k<num_atom_a;k++){
    sum+=local_listc[k];
}

*curr_count+=sum;

sum=0;
//Collect Grid Counts Across Blocks
for (t=0;t<blocks;t++){
	for (r=minn_x;r<maxx_x+1;r++){
		for (rr=minn_y;rr<maxx_y+1;rr++){
			for (rrr=minn_z;rrr<maxx_z+1;rrr++){
				n_of_grid=(r-minn_x)+((rr-minn_y)*x_side)+((rrr-minn_z)*x_side*y_side)+(t*x_side*y_side*z_side);
				n_of_grid2=(r-minn_x)+((rr-minn_y)*x_side)+((rrr-minn_z)*x_side*y_side);
				local_space2[n_of_grid2]+=local_space[n_of_grid];
			}
		}
	}
}

//Update Grid Array
for (r=minn_x;r<maxx_x+1;r++){
	for (rr=minn_y;rr<maxx_y+1;rr++){
		for (rrr=minn_z;rrr<maxx_z+1;rrr++){
			n_of_grid=(r-minn_x)+((rr-minn_y)*x_side)+((rrr-minn_z)*x_side*y_side);
                        grid_array[n_of_grid]+=local_space2[n_of_grid];
		}
	}
}

///////////////////////////////////////////////////////

free(local_lista);
free(local_listb);
free(local_listc);
free(local_x);
free(local_y);
free(local_z);
free(local_conversion);
free(local_space);
free(local_space2);

}//Main
