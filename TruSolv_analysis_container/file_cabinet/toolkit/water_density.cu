#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define CHECK(call) {   const hipError_t error = call; if (error != hipSuccess) { printf("Error: %s:%d, ", __FILE__, __LINE__); printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); exit(1); }}

__global__ void filldensity (int *a,int na,float *xx,float *yy,float *zz,int x_min,int y_min,int z_min,int x_max,int y_max,int z_max,int x_side,int y_side,int z_side,float *d_field,int num_points,float *atom_radius) {

        int k=threadIdx.x + blockDim.x * blockIdx.x;
        int j,kk,R,xxx,yyy,zzz;
        float curr_dist,radius,dimensionality;
        float pi;

        pi=3.1415926535897931;

	if(k<num_points){
		//Convert Point to XYZ Coords
		zzz=k/(x_side*y_side);
		R=k%(x_side*y_side);
		yyy=R/x_side;
		xxx=R%x_side;
		xxx=xxx+x_min;
		yyy=yyy+y_min;
		zzz=zzz+z_min;
		for(j=0;j<na;j++){
			kk=a[j]-1;
			curr_dist=sqrtf(((xx[kk]-float(xxx))*(xx[kk]-float(xxx)))+((yy[kk]-float(yyy))*(yy[kk]-float(yyy)))+((zz[kk]-float(zzz))*(zz[kk]-float(zzz))));
			if(curr_dist <= 6.0){
				radius=powf(((atom_radius[kk])/2.0),2.0);
				dimensionality=(-3.0/2.0);
				d_field[k]+=powf((2.0*pi*radius),dimensionality)*(expf((-1.0*curr_dist*curr_dist)/(2.0*radius)));
			}
		}
	}

} // End of Global

extern "C" void water_wrapper_2_(int *frame_count, double *x,double *y,double *z,int *min_x,int *min_y,int *min_z,int *max_x, int *max_y, int *max_z,int *sidex,int *sidey,int *sidez,double *grid_array,int *natoma,int *lista,int *natim,double *radius_array)
{//main

int k,r,rr,rrr,blocks,threads;
long int num_points,n_of_grid;
int devCount;
int num_atom_a=*natoma;
int n_atim=*natim;
int minn_x=*min_x;
int minn_y=*min_y;
int minn_z=*min_z;
int maxx_x=*max_x;
int maxx_y=*max_y;
int maxx_z=*max_z;
int x_side=*sidex;
int y_side=*sidey;
int z_side=*sidez;
int *local_lista;
float *local_density_field;
float *local_x, *local_y, *local_z;
float *local_radius;
int *dev_lista;
float *dev_density_field;
float *dev_x,*dev_y,*dev_z;
float *dev_radius;

//CHECK (cudaSetDevice ( 0 ) );

/////////////////////////////////////////////////////////////////

hipGetDeviceCount(&devCount);
//printf("CUDA Device Query...\n");
//printf("There are %d CUDA devices.\n", devCount);

// Iterate through devices
for (int i = 0; i < devCount; ++i){
        // Get device properties
        //printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        //printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
        threads=devProp.maxThreadsPerBlock;
}

blocks=0;
num_points=0;
num_points=x_side*y_side*z_side;

blocks=ceil(float(num_points)/float(threads))+1;

local_lista=(int *)malloc(num_atom_a*sizeof(int));
if(local_lista == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_x=(float *)malloc(n_atim*sizeof(float));
if(local_x == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_y=(float *)malloc(n_atim*sizeof(float));
if(local_y == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_z=(float *)malloc(n_atim*sizeof(float));
if(local_z == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}


local_radius=(float *)malloc(n_atim*sizeof(float));
if(local_radius == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}

local_density_field=(float *)malloc(num_points*sizeof(float));
if(local_density_field == NULL){
   printf("Error: %s:%d, ", __FILE__, __LINE__);
   exit(1);}


memset(local_density_field,0,num_points*sizeof(float));
///////////////////////////////////////////////////////


// Transfer Pointers from Fortran to Local C Arrays
for (k=0;k<num_atom_a;k++){
     local_lista[k]=lista[k];
}

for (k=0;k<n_atim;k++){
     local_x[k]=x[k];
     local_y[k]=y[k];
     local_z[k]=z[k];
     local_radius[k]=radius_array[k];
}

///////////////////////////////////////////////////////

CHECK (hipMalloc((void **) &dev_lista, num_atom_a*sizeof(int)) );
CHECK (hipMalloc((void **) &dev_density_field, num_points*sizeof(float)) );
CHECK (hipMalloc((void **) &dev_x, n_atim*sizeof(float)) );
CHECK (hipMalloc((void **) &dev_y, n_atim*sizeof(float)) );
CHECK (hipMalloc((void **) &dev_z, n_atim*sizeof(float)) );
CHECK (hipMalloc((void **) &dev_radius, n_atim*sizeof(float)) );


CHECK (hipMemcpy(dev_x, local_x, n_atim*sizeof(float), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_y, local_y, n_atim*sizeof(float), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_z, local_z, n_atim*sizeof(float), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_radius, local_radius, n_atim*sizeof(float), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_lista, local_lista, num_atom_a*sizeof(int), hipMemcpyHostToDevice) );
CHECK (hipMemcpy(dev_density_field, local_density_field, num_points*sizeof(float), hipMemcpyHostToDevice) );

///////////////////////////////////////////////////////

//printf("Launch!\n");
//printf("Blocks=%i\n",blocks);
//printf("Threads=%i\n",threads);


filldensity<<<blocks,threads>>>(dev_lista,num_atom_a,dev_x,dev_y,dev_z,minn_x,minn_y,minn_z,maxx_x,maxx_y,maxx_z,x_side,y_side,z_side,dev_density_field,num_points,dev_radius);

CHECK (hipMemcpy(local_density_field, dev_density_field, num_points*sizeof(float), hipMemcpyDeviceToHost) );

CHECK (hipFree(dev_lista) ); 
CHECK (hipFree(dev_x) ); 
CHECK (hipFree(dev_y) ); 
CHECK (hipFree(dev_z) ); 
CHECK (hipFree(dev_radius) ); 
CHECK (hipFree(dev_density_field) );
CHECK (hipDeviceReset());

///////////////////////////////////////////////////////

n_of_grid=0;
//Update Grid Array
for (r=minn_x;r<maxx_x+1;r++){
	for (rr=minn_y;rr<maxx_y+1;rr++){
		for (rrr=minn_z;rrr<maxx_z+1;rrr++){
			n_of_grid=(r-minn_x)+((rr-minn_y)*x_side)+((rrr-minn_z)*x_side*y_side);
                        grid_array[n_of_grid]+=local_density_field[n_of_grid];
		}
	}
}

///////////////////////////////////////////////////////

free(local_lista);
free(local_x);
free(local_y);
free(local_z);
free(local_radius);
free(local_density_field);

}//Main
